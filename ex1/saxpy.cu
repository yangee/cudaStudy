#include "hip/hip_runtime.h"
#include <stdio.h>

// Device code
__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
// Host Code
  float *x, *y, *d_x, *d_y; 
// x,y Points to the host arrays - d_x, d_y to the device arrays.

// malloc
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
// hipMalloc
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

// init host arr
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
// last option - direction of copy
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // thread blocks required to process all N elements of the arrays
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  // Launching grid of thread blocks - number of thread blocks in grid, number of threads in a thread block
  //

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
